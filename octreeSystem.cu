#include "hip/hip_runtime.h"
#include "octreeSystem.cuh"
#include <stdio.h>

// Constructor
octreeSystem::octreeSystem()
{

}

// Destructor
octreeSystem::~octreeSystem()
{

}

void octreeSystem::reset()
{
    // initialize octreeSystem
    octantList.clear();
    bucketCounter = 0;

    // Initialize the parent oct's cell
    octRectangle parentRect;
    parentRect.topLeft.x = 0;
    parentRect.topLeft.y = HEIGHT;
    parentRect.height = HEIGHT;
    parentRect.width = WIDTH;
    parentRect.calculateVertices();

    // Initialize the parent oct
    octant parentOct;
    parentOct.index = 0;
    parentOct.octRect = parentRect;
    parentOct.isBucket = true;
    parentOct.containedParticlesIndices.clear();

    // Reserve enough space for a fixed number of octants. This is required since dynamically allocating the
    // vector from a void inside one of the vector's contained objects will change memory addresses to the contained object,
    // The problem is that C++ will still use the old memory address for obtaining variables stored by that object.
    octantList.reserve(OCT_LIST_ALLOC);
    octantList.push_back(parentOct);
}

void octreeSystem::makeOctree(particleSystem& pSystem)
{
    // set parent oct's contained particles to 1..N
    for (int i = 0; i < N; i++)
    {
        bool withinWidth = (octantList[0].octRect.topLeft.x < pSystem.pos[i].x && pSystem.pos[i].x < octantList[0].octRect.topRight.x);
        bool withinHeight = (octantList[0].octRect.bottomLeft.y < pSystem.pos[i].y && pSystem.pos[i].y < octantList[0].octRect.topLeft.y);
        if (withinWidth && withinHeight)
            octantList[0].containedParticlesIndices.push_back(i);
    }

    // Divide the first octant
    if (octantList[0].containedParticlesIndices.size() > MAX_PARTICLES_PER_BUCKET)
    {
        octantList[0].isBucket = false;
        octantList[0].divide(octantList, pSystem, bucketCounter);
    }
}

// Arranges the particles in pSystem into a morton curve to send to GPU
void octreeSystem::arrangeAndTransfer(particleSystem& pSystem, deviceOctant* d_octantList, deviceParticle* d_deviceParticleList)
{
    pSystem.h_deviceParticleList = new deviceParticle[N];
    h_octantList = new deviceOctant[octCount];

    int MortonCounter = 0;
    for (int i = 0; i < octCount; i++)
    {
        int containedParticlesCount = octantList[i].containedParticlesIndices.size();

        // The neighbour search radius for this octant should be roughly proportional to this octant's size.
        // This is because extremely large octants are likely in less dense areas and require a larger search radius, and vice versa
        // Note: hCell should be an overestimate rather than an underestimate for accurate SPH
        float hCell = 2 * std::max(octantList[i].octRect.width, octantList[i].octRect.height) + 10;
        h_octantList[i].neibSearchRadius = hCell;

        // Transfers info storing the # of contained particles as well as # of neib buckets for this bucket
        h_octantList[i].containedParticleCount = containedParticlesCount;
        h_octantList[i].neibBucketCount = octantList[i].neibBucketIndices.size();

        // Allocate and transfer contained particle indices to the GPU
        h_octantList[i].firstContainedParticleIdx = MortonCounter;

        // Allocate and transfer neib bucket indices to the GPU
        hipMalloc((void**) &(h_octantList[i].d_neibBucketsIndices), h_octantList[i].neibBucketCount * sizeof(int));
        hipMemcpy(h_octantList[i].d_neibBucketsIndices, &(octantList[i].neibBucketIndices[0]), h_octantList[i].neibBucketCount * sizeof(int), hipMemcpyHostToDevice);

        for (int j = 0; j < containedParticlesCount; j++)
        {
            // Convert particle info into congruent arrays
            deviceParticle h_particle;
            int particleIdx = octantList[i].containedParticlesIndices[j];
            h_particle.particleData[0] = pSystem.pos[particleIdx].x;
            h_particle.particleData[1] = pSystem.pos[particleIdx].y;
            h_particle.particleData[2] = pSystem.pos[particleIdx].z;
            h_particle.particleData[3] = particleIdx;
            h_particle.particleData[4] = pSystem.prevpos[particleIdx].x;
            h_particle.particleData[5] = pSystem.prevpos[particleIdx].y;
            h_particle.particleData[6] = pSystem.prevpos[particleIdx].z;
            h_particle.particleData[7] = pSystem.mass[particleIdx];
            h_particle.particleData[8] = pSystem.smoothingLengths[particleIdx];
            h_particle.particleData[9] = pSystem.densities[particleIdx];
            h_particle.particleData[10] = pSystem.omegas[particleIdx];
            h_particle.particleData[11] = pSystem.pressures[particleIdx];
            pSystem.h_deviceParticleList[MortonCounter] = h_particle;
            MortonCounter++;
        }
    }

    // OctantList data is on GPU, now transfer the "meta data" to the GPU
    hipMemcpy(d_octantList, h_octantList, octCount*sizeof(deviceOctant), hipMemcpyHostToDevice);
    // Transfer rearranged particle data to GPU
    hipMemcpy(d_deviceParticleList, pSystem.h_deviceParticleList, N*sizeof(deviceParticle), hipMemcpyHostToDevice);
}

// Converts each bucket octant into a deviceOctant and sends it to the device.
/*deviceOctant* octreeSystem::sendToGPU()
{
    // Allocate device octant on device and record their "meta data" (pointers) on the CPU
    h_octantList = new deviceOctant[octCount];
    deviceOctant* d_octantList;

    for (int i = 0; i < octCount; i++)
    {

    }

    // OctantList data is on GPU, now transfer the "meta data" to the GPU
    hipMalloc((void**) &d_octantList, octCount*sizeof(deviceOctant));
    hipMemcpy(d_octantList, h_octantList, octCount*sizeof(deviceOctant), hipMemcpyHostToDevice);

    return d_octantList;
}*/

void octreeSystem::freeFromGPU()
{
    // Free dyanmically allocated memory
    for (int i = 0; i < octCount; i++)
    {
        hipFree(h_octantList[i].d_neibBucketsIndices);
    }

    delete[] h_octantList;
}


void octreeSystem::eliminateBranches()
{
    for (int i = 0; i < octantList.size(); i+=0)
    {
        if (octantList[i].isBucket)
            i++;
        else
            octantList.erase(octantList.begin() + i);
    }

    octCount = octantList.size();
}

// Find neighbour buckets for all bucket octants. Complexity is O(N*log(N))
void octreeSystem::findAllBucketNeibs()
{
    for (int i = 0; i < octantList.size(); i++)
    {
        if (octantList[i].isBucket)
            octantList[i].neibSearchTraversal(octantList, 0);
    }
}

// Constructor
octant::octant()
{

}

// Destructor
octant::~octant()
{

}

void octant::divide(std::vector<octant>& octantList, particleSystem& pSystem, int& bucketCounter)
{
    // Top left
    octant topLeftChild;
    //topLeftChild.parentIndex = index;
    topLeftChild.octRect.topLeft = octRect.topLeft;
    topLeftChild.octRect.width = octRect.width / 2.00f;
    topLeftChild.octRect.height = octRect.height / 2.00f;
    topLeftChild.octRect.calculateVertices();
    // Top right
    octant topRightChild;
    //topRightChild.parentIndex = index;
    topRightChild.octRect.topLeft.x = octRect.centre.x;
    topRightChild.octRect.topLeft.y = octRect.topRight.y;
    topRightChild.octRect.width = octRect.width / 2.00f;
    topRightChild.octRect.height = octRect.height / 2.00f;
    topRightChild.octRect.calculateVertices();
    // Bottom left
    octant bottomLeftChild;
    //bottomLeftChild.parentIndex = index;
    bottomLeftChild.octRect.topLeft.x = octRect.topLeft.x;
    bottomLeftChild.octRect.topLeft.y =  octRect.centre.y;
    bottomLeftChild.octRect.width = octRect.width / 2.00f;
    bottomLeftChild.octRect.height = octRect.height / 2.00f;
    bottomLeftChild.octRect.calculateVertices();
    // Bottom right
    octant bottomRightChild;
    //bottomRightChild.parentIndex = index;
    bottomRightChild.octRect.topLeft = octRect.centre;
    bottomRightChild.octRect.width = octRect.width / 2.00f;
    bottomRightChild.octRect.height = octRect.height / 2.00f;
    bottomRightChild.octRect.calculateVertices();

    // Assign particles to each new child oct
    for (int i = 0; i < containedParticlesIndices.size(); i++)
    {
        bool withinTopHalf =  pSystem.pos[containedParticlesIndices[i]].y > octRect.centre.y;
        bool withinLeftHalf =  pSystem.pos[containedParticlesIndices[i]].x < octRect.centre.x;

        if (withinTopHalf && withinLeftHalf)
            topLeftChild.containedParticlesIndices.push_back(containedParticlesIndices[i]);
        else if (withinTopHalf)
            topRightChild.containedParticlesIndices.push_back(containedParticlesIndices[i]);
        else if (withinLeftHalf)
            bottomLeftChild.containedParticlesIndices.push_back(containedParticlesIndices[i]);
        else
            bottomRightChild.containedParticlesIndices.push_back(containedParticlesIndices[i]);
    }

    // Top left child
    // If this child is a bucket, set its bucket index and assign this bucket index to its contained particles
    if (topLeftChild.containedParticlesIndices.size() <= MAX_PARTICLES_PER_BUCKET)
        topLeftChild.bucketIndex = bucketCounter++;
    topLeftChild.index = octantList.size();
    childrenIndices[0] = topLeftChild.index;
    octantList.push_back(topLeftChild);
    if (topLeftChild.containedParticlesIndices.size() > MAX_PARTICLES_PER_BUCKET)
        octantList[topLeftChild.index].divide(octantList, pSystem, bucketCounter);

    // Top right child
    // If this child is a bucket, set its bucket index and assign this bucket index to its contained particles
    if (topRightChild.containedParticlesIndices.size() <= MAX_PARTICLES_PER_BUCKET)
        topRightChild.bucketIndex = bucketCounter++;
    topRightChild.index = octantList.size();
    childrenIndices[1] = topRightChild.index;
    octantList.push_back(topRightChild);
    if (topRightChild.containedParticlesIndices.size() > MAX_PARTICLES_PER_BUCKET)
        octantList[topRightChild.index].divide(octantList, pSystem, bucketCounter);

    // Bottom left child
    // If this child is a bucket, set its bucket index and assign this bucket index to its contained particles
    if (bottomLeftChild.containedParticlesIndices.size() <= MAX_PARTICLES_PER_BUCKET)
        bottomLeftChild.bucketIndex = bucketCounter++;
    bottomLeftChild.index = octantList.size();
    childrenIndices[2] = bottomLeftChild.index;
    octantList.push_back(bottomLeftChild);
    if (bottomLeftChild.containedParticlesIndices.size() > MAX_PARTICLES_PER_BUCKET)
        octantList[bottomLeftChild.index].divide(octantList, pSystem, bucketCounter);

    // Bottom right child
    // If this child is a bucket, set its bucket index and assign this bucket index to its contained particles
    if (bottomRightChild.containedParticlesIndices.size() <= MAX_PARTICLES_PER_BUCKET)
        bottomRightChild.bucketIndex = bucketCounter++;
    bottomRightChild.index = octantList.size();
    childrenIndices[3] = bottomRightChild.index;
    octantList.push_back(bottomRightChild);
    if (bottomRightChild.containedParticlesIndices.size() > MAX_PARTICLES_PER_BUCKET)
        octantList[bottomRightChild.index].divide(octantList, pSystem, bucketCounter);

    // Since this oct has been divided, it can no longer be a bucket
    isBucket = false;
}

// Traverses octants recursively to check if they are within H_CELL. If they are, store them in neibBucketIndices
void octant::neibSearchTraversal(std::vector<octant>& octantList, int currentIndex)
{
    octant currentOct = octantList[currentIndex];
    float hCell = 2 * std::max(octRect.width, octRect.height) + 10;
    if (octRect.withinDistance(currentOct.octRect, hCell))
    {
        if (currentOct.isBucket)
            neibBucketIndices.push_back(currentOct.bucketIndex);
        else
        {
            for (int i = 0; i < 4; i++)
                neibSearchTraversal(octantList, currentOct.childrenIndices[i]);
        }
    }
}
