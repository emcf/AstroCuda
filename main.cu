// This is an octree-based GPU SPH demo
// Execute the build.sh script to compile and run the code.
// Written by Emmett McFarlane

#include "octreeSystem.cuh"
#include "particleSystem.cuh"
#include "physicsSystemGPU.cuh"
#include "displayHandler.cuh"
#include <GL/glut.h>
#include <time.h>

static octreeSystem octSystem;
static particleSystem pSystem;
static physicsSystemGPU sphSystem;
static displayHandler display;

void draw()
{
    glClear(GL_COLOR_BUFFER_BIT);
    display.drawMortonCurve(octSystem, pSystem);
    display.drawSmoothingLenghs(pSystem);
    for (int i = 0; i < octSystem.octCount; i++)
        display.drawOct(octSystem, i);
    for (int i = 0; i < N; i++)
        display.drawParticle(pSystem, i);
    glFlush();
}

int main(int argc, char* argv[])
{
    pSystem.init();
    display.init();
    glutDisplayFunc(draw);

    while (true)
    {
        // Build octree
        octSystem.reset();
        octSystem.makeOctree(pSystem);
        octSystem.findAllBucketNeibs();
        octSystem.eliminateBranches();

        // Rearrange particle data to work on GPU, send it to GPU
        deviceParticle* d_deviceParticleList;
        deviceOctant* d_octantList;
        hipMalloc((void**) &d_deviceParticleList, N*sizeof(deviceParticle));
        hipMalloc((void**) &d_octantList, octSystem.octCount*sizeof(deviceOctant));
        octSystem.arrangeAndTransfer(pSystem, d_octantList, d_deviceParticleList);

        // Compute density and integrate positions on GPU
        sphSystem.RunGPUSPH(octSystem, d_octantList, d_deviceParticleList);

        // Get data from GPU, rearrange particle data to work on host
        octSystem.freeFromGPU();
        pSystem.getFromGPU(d_deviceParticleList);

        draw();
    }
}
